#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdint.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <mutex>
#include <vector>
#include <sys/time.h>
#include <condition_variable>
#include "GPU.h"

#include "../stdexcept.hpp"


#define BAND_SIZE 32 
#define LOG_BLOCK_SIZE 7

#define BLOCK_SIZE (1 << LOG_BLOCK_SIZE)

int NUM_BLOCKS;
int NUM_DEVICES;
size_t BATCH_SIZE;

std::mutex mu;
std::condition_variable cv;
std::vector<int> available_gpus;

uint32_t num_unique_markers;

uint32_t** d_alignments;
float** d_score;
uint32_t** d_score_pos;
uint32_t** d_num_traceback;
uint64_t** d_common_markers;
uint32_t** d_num_common_markers;
uint64_t** d_batch_rid_markers;
uint64_t** d_rid_marker_pos;

using namespace shasta;


__global__
void initialize_batch_rid_markers (uint64_t* batch_rid_markers, uint32_t num_unique_markers, uint32_t batch_size) {
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    for (uint64_t i=bx; i < 2*batch_size; i+=gs) {
        uint64_t v, val;
        v = (i << (32+SHASTA_LOG_MAX_MARKERS_PER_READ));
        for (uint64_t j=tx; j< num_unique_markers; j+=bs) {
            val = v + (j << SHASTA_LOG_MAX_MARKERS_PER_READ);
            batch_rid_markers[i*num_unique_markers+j] = val;
        }
    }
    if (bx==0) {
        if (tx == 0) {
            uint64_t v = 2*batch_size;
            batch_rid_markers[2*batch_size*num_unique_markers] = (v << (32+SHASTA_LOG_MAX_MARKERS_PER_READ));
        }
    }
}

__global__
void skip_high_frequency_markers (uint64_t maxMarkerFrequency, uint32_t num_unique_markers, uint64_t* index_table, uint64_t* rid_marker_pos, uint64_t* sorted_rid_marker_pos, uint16_t* adjusted_pos) {
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    
    __shared__ uint64_t s, e;
    
    uint16_t sum = 0;
    uint16_t prev_sum = 0;
    uint64_t m_mask = ((uint64_t) 1 << 32) - 1;

    if (tx == 0) {
        s = index_table[bx*num_unique_markers];
        e = index_table[(bx+1)*num_unique_markers];
    }
    __syncthreads();

    for (uint64_t i = s; i < e; i+=bs) {
        uint64_t idx = i+tx;
        uint64_t v = rid_marker_pos[idx];
        uint64_t marker = ((v >> SHASTA_LOG_MAX_MARKERS_PER_READ) & m_mask);
        uint64_t sm=0, em=0;

        sum = prev_sum;
        if (idx < e) {
            uint64_t v = rid_marker_pos[idx];
            marker = ((v >> SHASTA_LOG_MAX_MARKERS_PER_READ) & m_mask);

            sm = index_table[bx*num_unique_markers+marker];
            em = index_table[bx*num_unique_markers+marker+1];
            if ((em-sm) <= maxMarkerFrequency) {
                sum += 1;
            }
        }

        __syncthreads();

        for (int s = 1; s <= bs; s *= 2) {
            int val = __shfl_up_sync(0xffffffff, sum, s, bs);

            if (tx >= s) {
                sum += val;
            }
        }
            
        __syncthreads();

        if (idx < e) {
            adjusted_pos[idx] = sum;
        }
        
        int prev = __shfl_down_sync(0xffffffff, sum, bs-1, bs);
        if (tx == 0) {
            prev_sum = prev;
        }

        __syncthreads();
    }
}

__global__
void find_common_markers (uint64_t maxMarkerFrequency, uint64_t n, uint32_t num_unique_markers, uint64_t* read_pairs, uint64_t* index_table, uint64_t* rid_marker_pos, uint64_t* sorted_rid_marker_pos, uint16_t* adjusted_pos, uint32_t* num_common_markers, uint64_t* common_markers)
{
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    uint64_t m_mask = ((uint64_t) 1 << 32) - 1;
    uint64_t p_mask = ((uint64_t) 1 << SHASTA_LOG_MAX_MARKERS_PER_READ) - 1;
    
    __shared__ uint32_t prefix[1+BLOCK_SIZE];

    __syncthreads();

    for (int i = bx; i < n; i+=gs) {
        if (tx == 0) {
            prefix[tx] = i*SHASTA_MAX_COMMON_MARKERS_PER_READ;
            num_common_markers[i] = 0;
        }
        __syncthreads();

        uint64_t v1 = read_pairs[2*i];
        uint64_t v2 = read_pairs[2*i+1];
        uint64_t rid1 = (v1 >> 32);
        uint64_t rid2 = (v2 >> 32);
        uint64_t l1 = ((v1 << 32) >> 32);
        uint64_t l2 = ((v2 << 32) >> 32);

        if ((l1 > 0) && (l2 > 0)) {
            uint64_t s1 = index_table[rid1*num_unique_markers];
            uint64_t s2 = index_table[rid2*num_unique_markers];
            uint64_t e2 = s2+l2;

            for (uint64_t j = s2; j < e2; j += bs) {
                uint64_t idx = tx+j;
                uint64_t marker;
                uint64_t sm1=0, sm2=0, em1=0, em2=0;

                prefix[1+tx] = 0; 

                if (idx < e2) {
                    uint64_t v = rid_marker_pos[idx];
                    marker = ((v >> SHASTA_LOG_MAX_MARKERS_PER_READ) & m_mask);

                    sm1 = index_table[rid1*num_unique_markers+marker];
                    em1 = index_table[rid1*num_unique_markers+marker+1];
                    sm2 = index_table[rid2*num_unique_markers+marker];
                    em2 = index_table[rid2*num_unique_markers+marker+1];

                    if ((em1 - sm1 <= maxMarkerFrequency) && (em2 - sm2 <= maxMarkerFrequency)) {
                        prefix[1+tx] = (em1-sm1);
                    }
                }

                __syncthreads();

                if (tx == 0) {
                    for (int r = 0; r < BLOCK_SIZE; r++) {
                        prefix[1+r] += prefix[r];
                    }
                }

                __syncthreads();

                uint32_t mhs = prefix[tx];
                uint32_t mhe = prefix[1+tx];
                
                __syncthreads();

                for (uint64_t k1 = 0; k1 < (mhe-mhs); k1++) {
                    if (mhs+k1 < (i+1)*SHASTA_MAX_COMMON_MARKERS_PER_READ) {
                        uint64_t sv1 = sorted_rid_marker_pos[sm1+k1];
                        uint64_t cm = (sv1 & p_mask);
                        uint64_t adj_pos1, adj_pos2;
                        
                        adj_pos1 = adjusted_pos[s1+cm];
                        adj_pos2 = adjusted_pos[idx]; 

                        cm = ((cm+1) << 16) + (1+idx-s2);
                        common_markers[mhs+k1] = cm + (adj_pos1 << 48) + (adj_pos2 << 32);
                    }
                }


                if (tx == 0) {
                    prefix[tx] = prefix[BLOCK_SIZE];
                }

                __syncthreads();
            }

            if (tx == 0) {
                uint32_t num_common = prefix[tx] - i*SHASTA_MAX_COMMON_MARKERS_PER_READ;
                num_common_markers[i] = num_common;
            }
        }

        __syncthreads();
    }
}

__global__
void find_traceback (int n, size_t maxSkip, size_t maxDrift, float* d_score, uint64_t const * __restrict__ d_common_markers, uint32_t const*  __restrict__ d_num_common_markers, uint32_t* d_score_pos, uint32_t* d_alignments, uint32_t* d_num_traceback, bool get_complete_traceback) {
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    float score;
    uint32_t score_pos;
    int num_common_markers;

    uint64_t p_mask = ((uint64_t) 1 << SHASTA_LOG_MAX_MARKERS_PER_READ) - 1;

    for (int i = bx; i < n; i += gs) {
        float max_score = 0;
        uint32_t max_score_pos = 0;
        uint32_t addr1 = i*SHASTA_MAX_COMMON_MARKERS_PER_READ;
        uint32_t addr2 = bx*SHASTA_MAX_COMMON_MARKERS_PER_READ;
        uint32_t addr3 = i*SHASTA_MAX_TB;
        if (!get_complete_traceback) {
            addr3 = 2*i;
        }

        num_common_markers = d_num_common_markers[i];
        if (num_common_markers >= SHASTA_MAX_COMMON_MARKERS_PER_READ) {
            num_common_markers = 0;
        }

        __syncthreads();

        for (int p = 0; p < num_common_markers; p++) {
//            uint64_t v =  __ldg(&d_common_markers[addr1+p]);
            uint64_t v =  d_common_markers[addr1+p];
            
            uint64_t l = ((v >> 32) & p_mask);
            uint64_t u = ((v >> 48) & p_mask);
            
            score = 0.01;
            score_pos = p;
            
            int ptr = p + tx;

            bool stop = false;
            __syncthreads();

            while (!stop) {
                ptr -= bs;
                if (ptr >= 0) {
                    uint64_t v1 = d_common_markers[addr1+ptr];
                    uint64_t l1, u1;
                    l1 = ((v1 >> 32) & p_mask);
                    u1 = ((v1 >> 48) & p_mask);
                    float a = l-l1;
                    float b = u-u1;
                    float alpha = fabs(a-b);
                    if ((l1 < l) && (u1 < u) && (u-u1 <= maxSkip) && (l-l1 <= maxSkip) && (alpha <= maxDrift)) {
                        float pscore = d_score[addr2+ptr]+1;
                        if (score < pscore) { 
                            score = pscore;
                            score_pos = ptr;
                        }
                    }

                    if (l > l1+maxSkip)  {
                        stop = true;
                    }
                }
                else {
                    stop = true;
                }
                
                stop = __shfl_sync(0xffffffff, stop, 0);
            }

            __syncthreads();

            // parallel reduction (max)
            for (int s = 1; s <= bs; s *= 2) {
                float val = __shfl_up_sync(0xffffffff, score, s, bs);
                uint32_t val_pos = __shfl_up_sync(0xffffffff, score_pos, s, bs);

                if (tx >= s) {
                    if (val > score) {
                        score = val;
                        score_pos = val_pos; 
                    }
                }
            }
            
            if (tx == bs-1) {
                d_score[addr2+p] = score;
                d_score_pos[addr2+p] = score_pos;
                if (score > max_score) {
                    max_score = score;
                    max_score_pos = p;
                }
            }
            __syncthreads();
        }

        __syncthreads();

        if (tx == bs-1) {
            int num_ptr = 0;
            uint64_t last_common_marker = 0;

            if (max_score > 0) {
                int curr_pos = max_score_pos;
                int prev_pos = max_score_pos + 1;

                while ((curr_pos >= 0) && (prev_pos > curr_pos)) {
                    prev_pos = curr_pos;
                    if (num_ptr < SHASTA_MAX_TB) {
                        if (get_complete_traceback) {
                            d_alignments[addr3+num_ptr] = d_common_markers[addr1+curr_pos];
                        }
                        else {
                            if (num_ptr == 0) {
                                d_alignments[addr3] = d_common_markers[addr1+curr_pos];
                            }
                            else {
                                last_common_marker = d_common_markers[addr1+curr_pos];
                            }
                        }
                    }
                    num_ptr++;
                    curr_pos = d_score_pos[addr2+curr_pos];
                }
            }

            if (num_ptr < SHASTA_MAX_TB) {
                if (get_complete_traceback) {
                    d_alignments[addr3+num_ptr] = 0;
                }
                else {
                    d_alignments[addr3+1] = last_common_marker;
                }
                d_num_traceback[i] = num_ptr;
            }
            else {
                d_alignments[addr3] = 0;
                d_num_traceback[i] = 0;
            }
        }
        __syncthreads();
    }
}

extern "C" std::tuple<int, size_t> shasta_initializeProcessors (size_t numUniqueMarkers) {
    int nDevices;

    num_unique_markers = (uint32_t) numUniqueMarkers;

    hipError_t err;
    
    err = hipGetDeviceCount(&nDevices);
    NUM_DEVICES = nDevices;

    if (err != hipSuccess) {
        throw runtime_error("GPU_ERROR: No GPU device found! Consider running without the --gpu flag.");
    }
    
    size_t device_memory;
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        device_memory = prop.totalGlobalMem;
        if (device_memory > 0xffffffff) {
            NUM_BLOCKS = (1 << 11);
            BATCH_SIZE = (1 << 12);
        }
        else {
            NUM_BLOCKS = (1 << 8);
            BATCH_SIZE = (1 << 9);
            break;
        }
        //printf("Device Number: %d\n", i);
        //printf("  Device name: %s\n", prop.name);
        //printf("  Memory Clock Rate (KHz): %d\n",
        //prop.memoryClockRate);
        //printf("  Memory Bus Width (bits): %d\n",
        //prop.memoryBusWidth);
        //printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
        //2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    d_alignments = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));

    d_score = (float**) malloc(nDevices*sizeof(float*));
    d_score_pos = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_num_traceback = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_common_markers = (uint64_t**) malloc(nDevices*sizeof(uint64_t*));
    d_num_common_markers = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_batch_rid_markers = (uint64_t**) malloc(nDevices*sizeof(uint64_t*));
    d_rid_marker_pos = (uint64_t**) malloc(nDevices*sizeof(uint64_t*));

    size_t num_bytes;

    for (int k=0; k<nDevices; k++) {
        
        available_gpus.push_back(k);

        err = hipSetDevice(k);
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: could not set device");
        }
        
        num_bytes = BATCH_SIZE*SHASTA_MAX_TB*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_alignments[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }

        num_bytes = NUM_BLOCKS*SHASTA_MAX_COMMON_MARKERS_PER_READ*sizeof(float);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_score[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }
        
        num_bytes = NUM_BLOCKS*SHASTA_MAX_COMMON_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_score_pos[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }
        
        num_bytes = BATCH_SIZE*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_num_traceback[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }
        
        num_bytes = BATCH_SIZE*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_num_common_markers[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }
        
        num_bytes = BATCH_SIZE*SHASTA_MAX_COMMON_MARKERS_PER_READ*sizeof(uint64_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_common_markers[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }

        num_bytes = (1+2*BATCH_SIZE*numUniqueMarkers)*sizeof(uint64_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_batch_rid_markers[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }

        num_bytes = (BATCH_SIZE*SHASTA_MAX_MARKERS_PER_READ)*sizeof(uint64_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_rid_marker_pos[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }

        initialize_batch_rid_markers<<<NUM_BLOCKS, BLOCK_SIZE>>> (d_batch_rid_markers[k], numUniqueMarkers, BATCH_SIZE);  
    }

    return std::make_tuple(nDevices, BATCH_SIZE);
}

extern "C" void shasta_alignBatchGPU (size_t maxMarkerFrequency, size_t maxSkip, size_t maxDrift, size_t n, uint64_t num_pos, uint64_t num_reads, uint64_t* batch_rid_marker_pos, uint64_t* batch_read_pairs, uint32_t* h_alignments, uint32_t* h_num_traceback, bool get_complete_traceback) {
    bool report_time = false;

    int k = -1;

    while (k < 0) {
        std::unique_lock<std::mutex> locker(mu);
        if (available_gpus.empty()) {
            cv.wait(locker, [](){return !available_gpus.empty();});
        }
        k = available_gpus.back();
        available_gpus.pop_back();
        locker.unlock();
    }

    struct timeval t1, t2, t3;
    long useconds, seconds, mseconds;
    
    hipError_t err; 

    err = hipSetDevice(k);
    if (err != hipSuccess) {
        throw runtime_error("GPU_ERROR: could not set device.\n");
    }
    
    gettimeofday(&t1, NULL);

    try {
        err = hipMemcpy(d_rid_marker_pos[k], batch_rid_marker_pos, num_pos*sizeof(uint64_t), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            throw runtime_error("Error: hipMemcpy failed!\n");
        }

        thrust::device_ptr<uint64_t> d_batch_rid_markers_ptr (d_batch_rid_markers[k]);
        thrust::device_ptr<uint64_t> d_rid_marker_pos_ptr (d_rid_marker_pos[k]);

        thrust::device_vector<uint64_t> t_d_rid_marker_pos (d_rid_marker_pos_ptr, d_rid_marker_pos_ptr + num_pos);
        thrust::device_vector<uint16_t> t_d_adjusted_pos (num_pos);
        thrust::device_vector<uint64_t> t_d_sorted_rid_marker_pos (num_pos);
        thrust::device_vector<uint64_t> t_d_rid_markers (d_batch_rid_markers_ptr, d_batch_rid_markers_ptr+num_reads*num_unique_markers+1);
        thrust::device_vector<uint64_t> t_d_read_pairs (batch_read_pairs, batch_read_pairs+2*n);
        thrust::device_vector<uint64_t> t_d_index_table (num_reads*num_unique_markers+1);

        thrust::copy (t_d_rid_marker_pos.begin(), t_d_rid_marker_pos.end(), t_d_sorted_rid_marker_pos.begin());
        thrust::sort(t_d_sorted_rid_marker_pos.begin(), t_d_sorted_rid_marker_pos.end());


        thrust::lower_bound(t_d_sorted_rid_marker_pos.begin(),
                t_d_sorted_rid_marker_pos.end(),
                t_d_rid_markers.begin(),
                t_d_rid_markers.end(),
                t_d_index_table.begin());

        gettimeofday(&t2, NULL);

        uint64_t* d_sorted_rid_marker_pos = thrust::raw_pointer_cast (t_d_sorted_rid_marker_pos.data());
        uint16_t* d_adjusted_pos = thrust::raw_pointer_cast (t_d_adjusted_pos.data()); 
        uint64_t* d_rid_marker_pos = thrust::raw_pointer_cast (t_d_rid_marker_pos.data());
        uint64_t* d_index_table = thrust::raw_pointer_cast (t_d_index_table.data());
        uint64_t* d_read_pairs = thrust::raw_pointer_cast (t_d_read_pairs.data());

        skip_high_frequency_markers <<< num_reads, 32>>> (maxMarkerFrequency, num_unique_markers, d_index_table, d_rid_marker_pos, d_sorted_rid_marker_pos, d_adjusted_pos);

        find_common_markers <<<NUM_BLOCKS, BLOCK_SIZE>>> (maxMarkerFrequency, n, num_unique_markers, d_read_pairs, d_index_table, d_rid_marker_pos, d_sorted_rid_marker_pos, d_adjusted_pos, d_num_common_markers[k], d_common_markers[k]);
        
        find_traceback <<<NUM_BLOCKS, BAND_SIZE>>>(n, maxSkip, maxDrift, d_score[k], d_common_markers[k], d_num_common_markers[k], d_score_pos[k], d_alignments[k], d_num_traceback[k], get_complete_traceback);

    }
    catch (std::bad_alloc) {
        throw runtime_error("Insufficient GPU memory. Try on GPU with larger memory or without --gpu option.\n");
    }

    err = hipMemcpy(h_num_traceback, d_num_traceback[k], n*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        throw runtime_error("Error: hipMemcpy failed!\n");
    }

    if (get_complete_traceback) {
        err = hipMemcpy(h_alignments, d_alignments[k], n*SHASTA_MAX_TB*sizeof(uint32_t), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            throw runtime_error("Error: hipMemcpy failed!\n");
        }
    }
    else {
        err = hipMemcpy(h_alignments, d_alignments[k], 2*n*sizeof(uint32_t), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            throw runtime_error("Error: hipMemcpy failed!\n");
        }
    }
    
    {
        std::unique_lock<std::mutex> locker(mu);
        available_gpus.push_back(k);
        locker.unlock();
        cv.notify_one();
    }
    
    gettimeofday(&t3, NULL);
    
    if (report_time) {
        useconds = t2.tv_usec - t1.tv_usec;
        seconds = t2.tv_sec - t1.tv_sec;
        mseconds = ((seconds) * 1000 + useconds/1000.0) + 0.5;
        fprintf(stderr, "Time elapsed (t2-t1): %ld msec \n", mseconds);

        useconds = t3.tv_usec - t1.tv_usec;
        seconds = t3.tv_sec - t1.tv_sec;
        mseconds = ((seconds) * 1000 + useconds/1000.0) + 0.5;
        fprintf(stderr, "Time elapsed (t3-t1): %ld msec \n", mseconds);
    }

    return;
}

extern "C" size_t shasta_getGpuBatchSize(){
    return BATCH_SIZE;
}

extern "C" void shasta_shutdownProcessors() {
    hipDeviceReset();
}
